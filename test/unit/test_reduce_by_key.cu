#include "hip/hip_runtime.h"
#include "../gtest.h"

#include <vector>

#include <backends/gpu/reduce_by_key.hpp>
#include <memory/memory.hpp>
#include <util/span.hpp>
#include <util/rangeutil.hpp>

using namespace arb;

template <typename T, typename I>
__global__
void reduce_kernel(const T* src, T* dst, const I* index, int n) {
    unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;

    unsigned mask = __ballot_sync(0xffffffff, tid<n);
    if (tid<n) {
        gpu::reduce_by_key(src[tid], dst, index[tid], mask);
    }
}

template <typename T>
std::vector<T> reduce(const std::vector<T>& in, size_t n_out, const std::vector<int>& index, unsigned block_dim=128) {
    EXPECT_EQ(in.size(), index.size());
    EXPECT_TRUE(std::is_sorted(index.begin(), index.end()));

    using array = memory::device_vector<T>;
    using iarray = memory::device_vector<int>;

    int n = in.size();

    array  src = memory::make_const_view(in);
    iarray idx = memory::make_const_view(index);
    array  dst(n_out, 0);

    unsigned grid_dim = (n-1)/block_dim + 1;
    reduce_kernel<<<grid_dim, block_dim>>>(src.data(), dst.data(), idx.data(), n);

    std::vector<T> out(n_out);
    memory::copy(dst, memory::make_view(out));

    return out;
}

TEST(reduce_by_key, no_repetitions)
{
    int n = 64;
    {
        std::vector<float> in(n, 1);
        std::vector<int> index = util::assign_from(util::make_span(0, n));

        auto out = reduce(in, n, index);
        for (auto o: out) EXPECT_EQ(o, 1.0f);
    }
    {
        std::vector<double> in(n, 1);
        std::vector<int> index = util::assign_from(util::make_span(0, n));

        auto out = reduce(in, n, index);
        for (auto o: out) EXPECT_EQ(o, 1.0);
    }
}

TEST(reduce_by_key, single_repeated_index)
{
    // Perform reduction of a sequence of 1s of length n
    // The expected result is n
    for (auto n: {1, 2, 7, 31, 32, 33, 63, 64, 65, 128}) {
        std::vector<double> in(n, 1);
        std::vector<int> index(n, 0);

        auto out = reduce(in, 1, index, 32);
        EXPECT_EQ(double(n), out[0]);
    }
    // Perform reduction of an ascending sequence of {1,2,3,...,n}
    // The expected result is n*(n+1)/2
    for (auto n: {1, 2, 7, 31, 32, 33, 63, 64, 65, 128}) {
        std::vector<double> in = util::assign_from(util::make_span(1, n+1));
        std::vector<int> index(n, 0);

        auto out = reduce(in, 1, index);
        EXPECT_EQ(out[0], double((n+1)*n/2));
    }
}

TEST(reduce_by_key, scatter)
{
    std::vector<int> index = {0,0,0,1,2,2,2,2,3,3,7,7,7,7,7,11};
    unsigned n = util::max_value(index)+1;
    std::vector<double> in(index.size(), 1);
    std::vector<double> expected = {3., 1., 4., 2., 0., 0., 0., 5., 0., 0., 0., 1.};

    unsigned m = index.size();

    EXPECT_EQ(n, expected.size());

    auto out = reduce(in, n, index);
    EXPECT_EQ(expected, out);

    // rerun with 7 threads per thread block, to test
    //  * using more than one thread block
    //  * thread blocks that are not a multiple of 32
    //  * thread blocks that are less than 32

    out = reduce(in, n, index, 7);
    EXPECT_EQ(expected, out);
}

template <typename I>
__global__
void key_set_pos_kernel(const I* index, int n, key_set_pos* out) {
    unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;

    unsigned mask = __ballot_sync(0xffffffff, tid<n);
    key_set_pos k(index[tid], mask);
    out[tid] = k;
}

std::vector<key_set_pos> run_key_set_pos(const std::vector<int>& index, unsigned block_dim=128) {
    using memory::device_vector;
    using memory::make_view;

    unsigned n = index.size();
    device_vector<int> index_gpu(memory::make_const_view(index));
    device_vector<key_set_pos> ks_gpu(n);

    unsigned grid_dim = (n-1)/block_dim + 1;

    std::vector<key_set_pos> ks(n);
    memory::copy(ks_gpu, memory::make_view(ks));

    return ks;
}

TEST(reduce_by_key, key_set_pos) {
    using ivector = std::vector<int>;
    using uvector = std::vector<unsigned>;

    {
	ivector index = { 2, 2, 2, 2, 3, 4, 4 };
	auto ks = run_key_set_pos(index);

	uvector expected_width = { 4, 3, 2, 1, 1, 2, 1 };
	uvector expected_is_root = { 1, 0, 0, 0, 1, 1, 0 };
	uvector expected_lane_id = { 0, 1, 2, 3, 4, 5, 6 };
	uvector expected_key_mask(7, 0x7fu);

	for (unsigned i = 0; i<7; ++i) {
	    EXPECT_EQ(expected_width[i], ks[i].width);
	    EXPECT_EQ(expected_lane_id[i], ks[i].lane_id);
	    EXPECT_EQ(expected_key_mask[i], ks[i].key_mask);
	    EXPECT_EQ(expected_is_root[i], ks[i].is_root);
	}
    }
}

// Test kernels that perform more than one reduction in a single invokation.
// Used to reproduce and test for synchronization issues on V100 GPUs.

template <typename T, typename I>
__global__
void reduce_twice_kernel(const T* src, T* dst, const I* index, int n) {
    unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;

    unsigned mask = __ballot_sync(0xffffffff, tid<n);
    if (tid<n) {
        gpu::reduce_by_key(src[tid], dst, index[tid], mask);
        gpu::reduce_by_key(src[tid], dst, index[tid], mask);
    }
}

template <typename T>
std::vector<T> reduce_twice(const std::vector<T>& in, size_t n_out, const std::vector<int>& index, unsigned block_dim=128) {
    EXPECT_EQ(in.size(), index.size());
    EXPECT_TRUE(std::is_sorted(index.begin(), index.end()));

    using array = memory::device_vector<T>;
    using iarray = memory::device_vector<int>;

    int n = in.size();

    array  src = memory::make_const_view(in);
    iarray idx = memory::make_const_view(index);
    array  dst(n_out, 0);

    unsigned grid_dim = (n-1)/block_dim + 1;
    reduce_twice_kernel<<<grid_dim, block_dim>>>(src.data(), dst.data(), idx.data(), n);

    std::vector<T> out(n_out);
    memory::copy(dst, memory::make_view(out));

    return out;
}

TEST(reduce_by_key, scatter_twice)
{
    std::vector<int> index = {0,0,0,1,2,2,3,7,7,7,11};
    unsigned n = util::max_value(index)+1;
    std::vector<double> in(index.size(), 1);
    std::vector<double> expected = {6., 2., 4., 2., 0., 0., 0., 6., 0., 0., 0., 2.};

    unsigned m = index.size();

    EXPECT_EQ(n, expected.size());

    auto out = reduce_twice(in, n, index);
    EXPECT_EQ(expected, out);

    // rerun with 7 threads per thread block, to test
    //  * using more than one thread block
    //  * thread blocks that are not a multiple of 32
    //  * thread blocks that are less than 32

    out = reduce_twice(in, n, index, 7);
    EXPECT_EQ(expected, out);
}
